#include "hip/hip_runtime.h"
#include <algorithm>

#include "caffe/common.hpp" 
#include "caffe/util/winograd.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe{

template <typename Dtype> 
__global__ void padSrc_gpu_kernel(const Dtype *src, Dtype *dst, int dataH, int dataW, int outH, int outW, int inputs, int batchs, int pad, float pData, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int highIdx = idx / (outH * outW); 
		int yIdx = (idx % (outH * outW)) / outW - pad;
		int xIdx = idx % outW - pad;

		if(xIdx < 0 || xIdx >= dataW || yIdx < 0 || yIdx >= dataH)
			dst[idx] = pData; 
		else
			dst[idx] = src[highIdx * dataH * dataW + yIdx * dataW + xIdx]; 
	}
}

template <typename Dtype> 
__global__ void winoWeight_gpu_kernel(const Dtype *src, Dtype *dst,  int inputs, int outputs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int outputIdx = idx / inputs;
		int inputIdx = idx % inputs;

		int gap = inputs * outputs;
		int kIdx = outputIdx * inputs * 9 + inputIdx * 9;
		int gIdx = idx % gap;


		dst[gIdx + 0] = src[kIdx + 0];
		dst[gIdx + gap] = ((src[kIdx + 0] + src[kIdx + 1] + src[kIdx + 2]) * 0.5);
		dst[gIdx + 2 * gap] = (src[kIdx + 0] - src[kIdx + 1] + src[kIdx + 2]) * 0.5;
		dst[gIdx + 3 * gap] = src[kIdx + 2];

		dst[gIdx + 4 * gap] = (src[kIdx + 0] + src[kIdx + 3] + src[kIdx + 6]) * 0.5 ;
		dst[gIdx + 5 * gap] = (src[kIdx + 0] + src[kIdx + 1] + src[kIdx + 2] + src[kIdx + 3] + src[kIdx + 4] + src[kIdx + 5] + src[kIdx + 6] + src[kIdx + 7] + src[kIdx + 8]) * 0.25;
		dst[gIdx + 6 * gap] = (src[kIdx + 0] - src[kIdx + 1] + src[kIdx + 2] + src[kIdx + 3] - src[kIdx + 4] + src[kIdx + 5] + src[kIdx + 6] - src[kIdx + 7] + src[kIdx + 8]) * 0.25;
		dst[gIdx + 7 * gap] = ( src[kIdx + 2] + src[kIdx + 5] + src[kIdx + 8]) * 0.5;

		dst[gIdx + 8 * gap] = ( src[kIdx + 0] - src[kIdx + 3] + src[kIdx + 6]) * 0.5;
		dst[gIdx + 9 * gap] =  (src[kIdx + 0] + src[kIdx + 1] + src[kIdx + 2] - src[kIdx + 3] - src[kIdx + 4] - src[kIdx + 5] + src[kIdx + 6] + src[kIdx + 7] + src[kIdx + 8]) * 0.25;
		dst[gIdx + 10 * gap] = (src[kIdx + 0] - src[kIdx + 1] + src[kIdx + 2] - src[kIdx + 3] + src[kIdx + 4] - src[kIdx + 5] + src[kIdx + 6] - src[kIdx + 7] + src[kIdx + 8]) * 0.25;
		dst[gIdx + 11 * gap] = ( src[kIdx + 2] - src[kIdx + 5] + src[kIdx + 8]) * 0.5;

		dst[gIdx + 12 * gap] = src[kIdx + 6];
		dst[gIdx + 13 * gap] = ( src[kIdx + 6] + src[kIdx + 7] + src[kIdx + 8]) * 0.5;
		dst[gIdx + 14 * gap] = ( src[kIdx + 6] - src[kIdx + 7] + src[kIdx + 8]) * 0.5;
		dst[gIdx + 15 * gap] = src[kIdx + 8];

	}
}

template <typename Dtype> 
__global__ void wino4x4Weight_gpu_kernel(const Dtype *src, Dtype *dst,  int inputs, int outputs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int outputIdx = idx / inputs;
		int inputIdx = idx % inputs;

		int gap = inputs * outputs;
		int kIdx = outputIdx * inputs * 9 + inputIdx * 9;
		int gIdx = idx % gap;


		//// -- project ---- ///




	}
}

template <typename Dtype> 
__global__ void wino4x4Weight_gpu_kernel(const Dtype *src, Dtype *dst,  int inputs, int outputs, int tNums, int zero_idx)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int outputIdx = idx / inputs;
		int inputIdx = idx % inputs;

		int gap = inputs * outputs;
		int kIdx = outputIdx * inputs * 9 + inputIdx * 9;
		int gIdx = idx % gap;


		//// -- project ---- ///




	}
}

template <typename Dtype> 
__global__ void wino6x6Weight_gpu_kernel(const Dtype *src, Dtype *dst,  int inputs, int outputs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int outputIdx = idx / inputs;
		int inputIdx = idx % inputs;

		int gap = inputs * outputs;
		int kIdx = outputIdx * inputs * 9 + inputIdx * 9;
		int gIdx = idx % gap;


		//// -- project ---- ///




	}
}

template <typename Dtype> 
__global__ void wino6x6Weight_gpu_kernel(const Dtype *src, Dtype *dst,  int inputs, int outputs, int tNums, int zero_idx)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int outputIdx = idx / inputs;
		int inputIdx = idx % inputs;

		int gap = inputs * outputs;
		int kIdx = outputIdx * inputs * 9 + inputIdx * 9;
		int gIdx = idx % gap;


		//// -- project ---- ///




	}
}

template <typename Dtype> 
__global__ void winoSrc_gpu_kernel(const Dtype *src, Dtype *dst, int dataH, int dataW, int tileH, int tileW, int inputs, int batchs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int gap = tileH * tileW * inputs * batchs;
		int batchIdx = idx / (tileH * tileW * inputs);
		int inputIdx = (idx / (tileH * tileW)) % inputs ;
		int yIdx = (idx % (tileH * tileW)) / tileW ;
		int xIdx = idx % tileW;
		int bIdx =  idx % gap;
		int sIdx = batchIdx * inputs * dataW * dataH + inputIdx * dataW * dataH + yIdx * dataW * 2 + xIdx * 2;

		dst[bIdx + 0] = src[sIdx + 0]  - src[sIdx + 2] - src[sIdx + 2 * dataW] + src[sIdx + 2 * dataW + 2];
		dst[bIdx + gap] = src[sIdx + 1]  + src[sIdx + 2] - src[sIdx + 2 * dataW + 1] - src[sIdx + 2 * dataW + 2];
		dst[bIdx + 2 * gap] = -1 * src[sIdx + 1] + src[sIdx + 2] + src[sIdx + 2 * dataW + 1] - src[sIdx + 2 * dataW + 2];
		dst[bIdx + 3 * gap] = src[sIdx + 1] - src[sIdx + 3] - src[sIdx + 2 * dataW + 1] + src[sIdx + 2 * dataW + 3];

		dst[bIdx + 4 * gap] = src[sIdx + dataW] - src[sIdx + dataW + 2] + src[sIdx + 2 * dataW] - src[sIdx + 2 * dataW + 2];
		dst[bIdx + 5 * gap] = src[sIdx + dataW + 1] + src[sIdx + dataW + 2] + src[sIdx + 2 * dataW + 1] + src[sIdx + 2 * dataW + 2];
		dst[bIdx + 6 * gap] = -1 * src[sIdx + dataW + 1] + src[sIdx + dataW + 2] - src[sIdx + 2 * dataW + 1] + src[sIdx + 2 * dataW + 2];
		dst[bIdx + 7 * gap] = src[sIdx + dataW + 1] - src[sIdx + dataW + 3] + src[sIdx + 2 * dataW + 1] - src[sIdx + 2 * dataW + 3];

		dst[bIdx + 8 * gap] = -1 * src[sIdx + dataW] + src[sIdx + dataW + 2] + src[sIdx + 2 * dataW] - src[sIdx + 2 * dataW + 2];
		dst[bIdx + 9 * gap]  = -1 * src[sIdx + dataW + 1] - src[sIdx + dataW + 2] + src[sIdx + 2 * dataW + 1] + src[sIdx + 2 * dataW + 2];
		dst[bIdx + 10 * gap] = src[sIdx + dataW + 1] - src[sIdx + dataW + 2] - src[sIdx + 2 * dataW + 1] + src[sIdx + 2 * dataW + 2];
		dst[bIdx + 11 * gap] = -1 * src[sIdx + dataW + 1] + src[sIdx + dataW + 3] + src[sIdx + 2 * dataW + 1] - src[sIdx + 2 * dataW + 3];

		dst[bIdx + 12 * gap] = src[sIdx + dataW] - src[sIdx + dataW + 2] - src[sIdx + 3 * dataW] + src[sIdx + 3 * dataW + 2];
		dst[bIdx + 13 * gap] = src[sIdx + dataW + 1] + src[sIdx + dataW + 2] - src[sIdx + 3 * dataW + 1] - src[sIdx + 3 * dataW + 2];
		dst[bIdx + 14 * gap] = -1 * src[sIdx + dataW + 1] + src[sIdx + dataW + 2] + src[sIdx + 3 * dataW + 1] - src[sIdx + 3 * dataW + 2];
		dst[bIdx + 15 * gap] = src[sIdx + dataW + 1] - src[sIdx + dataW + 3] - src[sIdx + 3 * dataW + 1] + src[sIdx + 3 * dataW + 3];
	}
}


template <typename Dtype> 
__global__ void wino4x4Src_gpu_kernel(const Dtype *src, Dtype *dst, int dataH, int dataW, int tileH, int tileW, int inputs, int batchs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int gap = tileH * tileW * inputs * batchs;
		int batchIdx = idx / (tileH * tileW * inputs);
		int inputIdx = (idx / (tileH * tileW)) % inputs ;
		int yIdx = (idx % (tileH * tileW)) / tileW ;
		int xIdx = idx % tileW;
		int bIdx =  idx % gap;
		int sIdx = batchIdx * inputs * dataW * dataH + inputIdx * dataW * dataH + yIdx * dataW * 2 + xIdx * 2;

		//// -- project ---- ///

	}
}

template <typename Dtype> 
__global__ void wino6x6Src_gpu_kernel(const Dtype *src, Dtype *dst, int dataH, int dataW, int tileH, int tileW, int inputs, int batchs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int gap = tileH * tileW * inputs * batchs;
		int batchIdx = idx / (tileH * tileW * inputs);
		int inputIdx = (idx / (tileH * tileW)) % inputs ;
		int yIdx = (idx % (tileH * tileW)) / tileW ;
		int xIdx = idx % tileW;
		int bIdx =  idx % gap;
		int sIdx = batchIdx * inputs * dataW * dataH + inputIdx * dataW * dataH + yIdx * dataW * 2 + xIdx * 2;

		//// -- project ---- ///

	}
}


template <typename Dtype> 
__global__ void winoSrcAddOpt_gpu_kernel(const Dtype *src, Dtype *dst, int dataH, int dataW, int tileH, int tileW, int inputs, int batchs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int gap = tileH * tileW * inputs * batchs;

		int batchIdx = idx / (tileH * tileW * inputs);
		int inputIdx = (idx / (tileH * tileW)) % inputs ;

		int yIdx = (idx % (tileH * tileW)) / tileW ;
		int xIdx = idx % tileW;

		int bIdx =  idx % gap;
		int sIdx = batchIdx * inputs * dataW * dataH + inputIdx * dataW * dataH + yIdx * dataW * 2 + xIdx * 2;

		float C[16]; 

		//// -- project ---- ///

	}
}

template <typename Dtype> 
__global__ void wino4x4SrcAddOpt_gpu_kernel(const Dtype *src, Dtype *dst, int dataH, int dataW, int tileH, int tileW, int inputs, int batchs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int gap = tileH * tileW * inputs * batchs;

		int batchIdx = idx / (tileH * tileW * inputs);
		int inputIdx = (idx / (tileH * tileW)) % inputs ;

		int yIdx = (idx % (tileH * tileW)) / tileW ;
		int xIdx = idx % tileW;

		int bIdx =  idx % gap;
		int sIdx = batchIdx * inputs * dataW * dataH + inputIdx * dataW * dataH + yIdx * dataW * 2 + xIdx * 2;


		//// -- project ---- ///

	}
}

template <typename Dtype> 
__global__ void wino6x6SrcAddOpt_gpu_kernel(const Dtype *src, Dtype *dst, int dataH, int dataW, int tileH, int tileW, int inputs, int batchs, int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int gap = tileH * tileW * inputs * batchs;

		int batchIdx = idx / (tileH * tileW * inputs);
		int inputIdx = (idx / (tileH * tileW)) % inputs ;

		int yIdx = (idx % (tileH * tileW)) / tileW ;
		int xIdx = idx % tileW;

		int bIdx =  idx % gap;
		int sIdx = batchIdx * inputs * dataW * dataH + inputIdx * dataW * dataH + yIdx * dataW * 2 + xIdx * 2;


		//// -- project ---- ///

	}
}



template <typename Dtype> 
__global__ void winoMulti_gpu_kernel(const Dtype *A, const Dtype *B, Dtype *C, int Ah, int Bw, int Aw, const float alpha, const float beta)
{
	int bx = blockIdx.x;
	int by = blockIdx.y; 
	int bz = blockIdx.z; 

	int tx = threadIdx.x; 
	int ty = threadIdx.y; 

	int aBegin = bz * Aw * Ah + Aw *32 * by; 
	int aEnd = Aw; 
	int aStep = 32; 

	int bBegin = bz * Bw * Aw + 32 * bx; 
	int bStep = 32; 

	float Csub = 0; 

	for(int a = 0, b = 0; a < aEnd; a += aStep, b+= bStep)
	{
		__shared__ float As[32][32]; 
		__shared__ float Bs[32][32]; 

		if( ((tx+a) < Aw) && ((32 * by + ty) < Ah))
			As[ty][tx] = A[aBegin + a + Aw * ty + tx]; 
		else 
			As[ty][tx] = 0; 

		if( ((32 * bx + tx) < Bw) && ( (b + ty) < Aw))
			Bs[ty][tx] = B[bBegin + Bw * (b + ty) + tx]; 
		else 
			Bs[ty][tx] = 0; 

		__syncthreads(); 

#pragma unroll
		for(int k = 0; k < 32; k++)
		{
			Csub += As[ty][k] * Bs[k][tx]; 
		}

		__syncthreads(); 
	}

	int cW = 32 * bx + tx; 
	int cH = 32 * by + ty;

	if((cW < Bw) && (cH < Ah))   
		C[bz * Bw * Ah + Bw * cH + cW] = Csub; 

}


template <typename Dtype> 
__global__ void winoDst_gpu_kernel(const Dtype *src, Dtype * dst, const int tileH, const int tileW, const int outH, const int outW, const int outputs, const int batchs, const int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int highIdx = idx / (tileW * tileH);
		int yIdx = (idx % (tileW * tileH)) / tileW;
		int xIdx = idx % tileW;
		int rIdx = highIdx * outW * outH + yIdx * outW * 2 + xIdx * 2;
		int mIdx = (idx % tNums); 
		int gap = batchs * outputs * tileH * tileW;

		float tmp ; 
		tmp = src[mIdx + gap * 0] + src[mIdx + gap * 1] + src[mIdx + gap * 2] + src[mIdx + gap * 4] + src[mIdx + gap * 5] + src[mIdx + gap * 6] + src[mIdx + gap * 8] + src[mIdx + gap * 9] + src[mIdx + gap * 10];
//		tmp = fabs(tmp) < 0.000001 ? 0 : tmp; 
//      dst[rIdx + 0] = bias[outIdx] + tmp; 
		dst[rIdx + 0] = tmp; 
               
      tmp = src[mIdx + gap * 1] - src[mIdx + gap * 2] - src[mIdx + gap * 3] + src[mIdx + gap * 5] - src[mIdx + gap * 6] - src[mIdx + gap * 7] + src[mIdx + gap * 9] - src[mIdx + gap * 10] - src[mIdx + gap * 11];
//		tmp = fabs(tmp) < 0.000001? 0 : tmp; 
//		dst[rIdx + 1] = bias[utIdx] + tmp;
		dst[rIdx + 1] = tmp;

		tmp = src[mIdx + gap *4] + src[mIdx + gap * 5] + src[mIdx + gap * 6] - src[mIdx + gap * 8] - src[mIdx + gap * 9] - src[mIdx + gap * 10] - src[mIdx + gap * 12] - src[mIdx + gap * 13] - src[mIdx + gap * 14];
//		tmp = fabs(tmp) < 0.00000 ? 0 : tmp; 
//		dst[rIdx + outW] = bias[outIdx] + tmp; 
		dst[rIdx + outW] =  tmp; 

		tmp = src[mIdx + gap * 5] - src[mIdx + gap * 6] - src[mIdx + gap * 7] - src[mIdx + gap * 9] + src[mIdx + gap * 10] + src[mIdx + gap * 11] - src[mIdx + gap * 13] + src[mIdx + gap * 14] + src[mIdx + gap * 15];
//		tmp = fabs(tmp) < 0.000001 ? 0 : tmp; 
//		dst[rIdx + outW + 1] = bias[outIdx] + tmp; 
		dst[rIdx + outW + 1] = tmp; 

	}
}

template <typename Dtype> 
__global__ void wino4x4Dst_gpu_kernel(const Dtype *src, Dtype * dst, const int tileH, const int tileW, const int outH, const int outW, const int outputs, const int batchs, const int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int highIdx = idx / (tileW * tileH);
		int yIdx = (idx % (tileW * tileH)) / tileW;
		int xIdx = idx % tileW;
		int rIdx = highIdx * outW * outH + yIdx * outW * 2 + xIdx * 2;
		int mIdx = (idx % tNums); 
		int gap = batchs * outputs * tileH * tileW;
			
						
		//// -- project ---- //		

	}
}

template <typename Dtype> 
__global__ void wino6x6Dst_gpu_kernel(const Dtype *src, Dtype * dst, const int tileH, const int tileW, const int outH, const int outW, const int outputs, const int batchs, const int tNums)
{

	CUDA_KERNEL_LOOP(idx, tNums) {
		int highIdx = idx / (tileW * tileH);
		int yIdx = (idx % (tileW * tileH)) / tileW;
		int xIdx = idx % tileW;
		int rIdx = highIdx * outW * outH + yIdx * outW * 2 + xIdx * 2;
		int mIdx = (idx % tNums); 
		int gap = batchs * outputs * tileH * tileW;
			
						
		//// -- project ---- //		

	}
}



template <typename Dtype> 
__global__ void winoDstAddOpt_gpu_kernel(const Dtype *src, Dtype * dst, const int tileH, const int tileW, const int outH, const int outW, const int outputs, const int batchs, const int tNums)
{


	CUDA_KERNEL_LOOP(idx, tNums) {
		
		int highIdx = idx / (tileW * tileH);

		int yIdx = (idx % (tileW * tileH)) / tileW;
		int xIdx = idx % tileW;

		int rIdx = highIdx * outW * outH + yIdx * outW * 2 + xIdx * 2;
		int mIdx = (idx % tNums); 
		int gap = batchs * outputs * tileH * tileW;

        float tmp; 
		float A[16]; 

		//// -- project ---- ///

	}


}


template <typename Dtype> 
__global__ void wino4x4DstAddOpt_gpu_kernel(const Dtype *src, Dtype * dst, const int tileH, const int tileW, const int outH, const int outW, const int outputs, const int batchs, const int tNums)
{


	CUDA_KERNEL_LOOP(idx, tNums) {
		
		int highIdx = idx / (tileW * tileH);

		int yIdx = (idx % (tileW * tileH)) / tileW;
		int xIdx = idx % tileW;

		int rIdx = highIdx * outW * outH + yIdx * outW * 2 + xIdx * 2;
		int mIdx = (idx % tNums); 
		int gap = batchs * outputs * tileH * tileW;


		//// -- project ---- ///

	}
}

template <typename Dtype> 
__global__ void wino6x6DstAddOpt_gpu_kernel(const Dtype *src, Dtype * dst, const int tileH, const int tileW, const int outH, const int outW, const int outputs, const int batchs, const int tNums)
{


	CUDA_KERNEL_LOOP(idx, tNums) {
		
		int highIdx = idx / (tileW * tileH);

		int yIdx = (idx % (tileW * tileH)) / tileW;
		int xIdx = idx % tileW;

		int rIdx = highIdx * outW * outH + yIdx * outW * 2 + xIdx * 2;
		int mIdx = (idx % tNums); 
		int gap = batchs * outputs * tileH * tileW;


		//// -- project ---- ///

	}
}


template <typename Dtype>
void winoWeight_gpu(const int num_inputs, const int num_outputs, 
					const Dtype *weight, Dtype *wino_weight, const int wino_tile_size )
{
	int num_kernels = num_inputs * num_outputs;

	if((wino_tile_size == 2) || (wino_tile_size == 12))
		winoWeight_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
			                         CAFFE_CUDA_NUM_THREADS>>>(weight, wino_weight, num_inputs, num_outputs, num_kernels); 
	else if((wino_tile_size == 4) || (wino_tile_size == 14))
		wino4x4Weight_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
			                         CAFFE_CUDA_NUM_THREADS>>>(weight, wino_weight, num_inputs, num_outputs, num_kernels); 
	else if((wino_tile_size == 6) || (wino_tile_size == 16))
		wino6x6Weight_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
			                         CAFFE_CUDA_NUM_THREADS>>>(weight, wino_weight, num_inputs, num_outputs, num_kernels); 

}

template void winoWeight_gpu<float>(const int num_inputs, const int num_outputs, 
									const float *weight, float *wino_weight, const int wino_tile_size); 
template void winoWeight_gpu<double>(const int num_inputs, const int num_outputs, 
									const double *weight, double *wino_weight, const int wino_tile_size); 




template <typename Dtype>
void padSrc_gpu(const int batchs, const int num_inputs, const int height, const int width, 
				const int height_pad, const int width_pad,
				const Dtype *input, Dtype *input_pad)
{

	int num_kernels = batchs * num_inputs * (height + height_pad * 2) * (width + width_pad * 2); 
	
	padSrc_gpu_kernel<Dtype><<< CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(input, input_pad, height, width, height + height_pad *2 , width + width_pad * 2, num_inputs, batchs, height_pad, 0, num_kernels); 

}

template void padSrc_gpu<float>(const int batchs, const int num_inputs, const int height, const int width, 
				const int height_pad, const int width_pad,
				const float *input, float *input_pad); 
template void padSrc_gpu<double>(const int batchs, const int num_inputs, const int height, const int width, 
				const int height_pad, const int width_pad,
				const double *input, double *input_pad); 


template <typename Dtype>
void winoSrc_gpu(const int batchs, const int num_inputs, const int tileH, const int tileW, 
				const int height, const int width, // include padding 
				const Dtype *m_matrix, Dtype *v_matrix, const int wino_tile_size)
{
	int num_kernels = batchs * num_inputs * tileH * tileW;

	if(wino_tile_size == 2)
	{
		winoSrc_gpu_kernel<Dtype><<< CAFFE_GET_BLOCKS(num_kernels),
				                         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, v_matrix, height, width,  tileH, tileW, num_inputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 12)
	{
		winoSrcAddOpt_gpu_kernel<Dtype><<< CAFFE_GET_BLOCKS(num_kernels),
			                         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, v_matrix, height, width,  tileH, tileW, num_inputs, batchs, num_kernels); 
	}
	else if (wino_tile_size == 4)
	{
		wino4x4Src_gpu_kernel<Dtype><<< CAFFE_GET_BLOCKS(num_kernels),
				                         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, v_matrix, height, width,  tileH, tileW, num_inputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 14)
	{
		winoSrcAddOpt_gpu_kernel<Dtype><<< CAFFE_GET_BLOCKS(num_kernels),
			                         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, v_matrix, height, width,  tileH, tileW, num_inputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 6)
	{
		wino6x6Src_gpu_kernel<Dtype><<< CAFFE_GET_BLOCKS(num_kernels),
				                         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, v_matrix, height, width,  tileH, tileW, num_inputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 16)
	{
		wino6x6SrcAddOpt_gpu_kernel<Dtype><<< CAFFE_GET_BLOCKS(num_kernels),
			                         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, v_matrix, height, width,  tileH, tileW, num_inputs, batchs, num_kernels); 
	}
}

template void winoSrc_gpu<float>(const int batchs, const int num_inputs, const int tileH, const int tileW, 
						const int height, const int width, // include padding 
						const float *m_matrix, float *v_matrix, const int wino_tile_size); 
template void winoSrc_gpu<double>(const int batchs, const int num_inputs, const int tileH, const int tileW, 
						const int height, const int width, // include padding 
						const double *m_matrix, double *v_matrix, const int wino_tile_size); 



template <typename Dtype>
void winoMulti_gpu(const int batchs, const int num_inputs, const int num_outputs, const int tileH, const int tileW, 
					const Dtype *u_matrix, Dtype *v_matrix, Dtype *m_matrix, const int wino_tile_size)
{

	int batched = (wino_tile_size + 2) * (wino_tile_size + 2); 


	for(int i = 0; i < batched; i++)
	{
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_outputs, batchs*tileH*tileW, num_inputs, (Dtype)1., u_matrix + i * num_inputs * num_outputs , v_matrix + i * tileH * tileW * num_inputs * batchs, (Dtype)0., m_matrix + i * batchs * num_outputs * tileH * tileW); 
	}


}

template void winoMulti_gpu<float>(const int batchs, const int num_inputs, const int num_outputs, const int tileH, const int tileW, 
									const float *u_matrix, float *v_matrix, float *m_matrix, const int wino_tile_size); 
template void winoMulti_gpu<double>(const int batchs, const int num_inputs, const int num_outputs, const int tileH, const int tileW, 
									const double *u_matrix, double *v_matrix, double *m_matrix, const int wino_tile_size); 




template <typename Dtype>
void winoDst_gpu(const int batchs, const int num_outputs, const int tileH, const int tileW, const int height, const int width,
				 Dtype *m_matrix, Dtype *output, const int wino_tile_size)
{
	
	int num_kernels = batchs * num_outputs * tileH * tileW;

	if(wino_tile_size == 2)
	{
		winoDst_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
					                 CAFFE_CUDA_NUM_THREADS>>>(m_matrix, output, tileH, tileW, height, width, num_outputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 12)
	{
		winoDstAddOpt_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
								         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, output, tileH, tileW, height, width, num_outputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 4)
	{
		wino4x4Dst_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
					                 CAFFE_CUDA_NUM_THREADS>>>(m_matrix, output, tileH, tileW, height, width, num_outputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 14)
	{
		wino4x4DstAddOpt_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
								         CAFFE_CUDA_NUM_THREADS>>>(m_matrix, output, tileH, tileW, height, width, num_outputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 6)
	{
		wino6x6Dst_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
					                 CAFFE_CUDA_NUM_THREADS>>>(m_matrix, output, tileH, tileW, height, width, num_outputs, batchs, num_kernels); 
	}
	else if(wino_tile_size == 16)
	{
		wino6x6DstAddOpt_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
					                 CAFFE_CUDA_NUM_THREADS>>>(m_matrix, output, tileH, tileW, height, width, num_outputs, batchs, num_kernels); 
	}
}

template void winoDst_gpu(const int batchs, const int num_outputs, const int tileH, const int tileW, const int height, const int width,
						 float *m_matrix, float *output, const int wino_tile_size); 

template void winoDst_gpu(const int batchs, const int num_outputs, const int tileH, const int tileW, const int height, const int width,
						 double *m_matrix, double *output, const int wino_tile_size); 

} // namespaece caffe 
